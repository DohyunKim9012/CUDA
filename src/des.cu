/*
 * Data Encryption Standard (DES) Implementation in C
 *
 * Reference: "FIPS Publication 46-3"
 *            (http://csrc.nist.gov/publications/fips/fips46-3/fips46-3.pdf)
 * Test case: J. Orlin Grabbe, "The DES Algorithm Illustrated"
 *            (https://www.uop.edu.jo/issa/isec/Des-Example.doc) *
 *
*/


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>
#include <string.h>
#include <unistd.h>
#include <sys/time.h>

#define CEIL(a, b) (((a) / (b)) + (((a) % (b)) > 0 ? 1 : 0))
#define MAX_THREADS 512
#define MAX_BLOCKS 65535
// ---------- HOST --------------
#define IP_SIZE 64
static const int hostIP[IP_SIZE] = {
  58, 50, 42, 34, 26, 18, 10,  2,
  60, 52, 44, 36, 28, 20, 12,  4,
  62, 54, 46, 38, 30, 22, 14,  6,
  64, 56, 48, 40, 32, 24, 16,  8,
  57, 49, 41, 33, 25, 17,  9,  1,
  59, 51, 43, 35, 27, 19, 11,  3,
  61, 53, 45, 37, 29, 21, 13,  5,
  63, 55, 47, 39, 31, 23, 15,  7
};

// ---------- HOST --------------
#define IP_1_SIZE 64
static const int hostIP_1[IP_1_SIZE] = {
  40,  8, 48, 16, 56, 24, 64, 32,
  39,  7, 47, 15, 55, 23, 63, 31,
  38,  6, 46, 14, 54, 22, 62, 30,
  37,  5, 45, 13, 53, 21, 61, 29,
  36,  4, 44, 12, 52, 20, 60, 28,
  35,  3, 43, 11, 51, 19, 59, 27,
  34,  2, 42, 10, 50, 18, 58, 26,
  33,  1, 41,  9, 49, 17, 57, 25
};

// ---------- HOST --------------
#define E_SIZE 48
static const int hostE[E_SIZE] = {
  32,  1,  2,  3,  4,  5,
   4,  5,  6,  7,  8,  9,
   8,  9, 10, 11, 12, 13,
  12, 13, 14, 15, 16, 17,
  16, 17, 18, 19, 20, 21,
  20, 21, 22, 23, 24, 25,
  24, 25, 26, 27, 28, 29,
  28, 29, 30, 31, 32,  1
};

// ---------- HOST -------------
#define S_SIZE 8
static const int hostS[S_SIZE][4][16] = {
  {
    { 14,  4, 13,  1,  2, 15, 11,  8,  3, 10,  6, 12,  5,  9,  0,  7 },
    {  0, 15,  7,  4, 14,  2, 13,  1, 10,  6, 12, 11,  9,  5,  3,  8 },
    {  4,  1, 14,  8, 13,  6,  2, 11, 15, 12,  9,  7,  3, 10,  5,  0 },
    { 15, 12,  8,  2,  4,  9,  1,  7,  5, 11,  3, 14, 10,  0,  6, 13 }
  },
  {
    { 15,  1,  8, 14,  6, 11,  3,  4,  9,  7,  2, 13, 12,  0,  5, 10 },
    {  3, 13,  4,  7, 15,  2,  8, 14, 12,  0,  1, 10,  6,  9, 11,  5 },
    {  0, 14,  7, 11, 10,  4, 13,  1,  5,  8, 12,  6,  9,  3,  2, 15 },
    { 13,  8, 10,  1,  3, 15,  4,  2, 11,  6,  7, 12,  0,  5, 14,  9 }
  },
  {
    { 10,  0,  9, 14,  6,  3, 15,  5,  1, 13, 12,  7, 11,  4,  2,  8 },
    { 13,  7,  0,  9,  3,  4,  6, 10,  2,  8,  5, 14, 12, 11, 15,  1 },
    { 13,  6,  4,  9,  8, 15,  3,  0, 11,  1,  2, 12,  5, 10, 14,  7 },
    {  1, 10, 13,  0,  6,  9,  8,  7,  4, 15, 14,  3, 11,  5,  2, 12 }
  },
  {
    {  7, 13, 14,  3,  0,  6,  9, 10,  1,  2,  8,  5, 11, 12,  4, 15 },
    { 13,  8, 11,  5,  6, 15,  0,  3,  4,  7,  2, 12,  1, 10, 14,  9 },
    { 10,  6,  9,  0, 12, 11,  7, 13, 15,  1,  3, 14,  5,  2,  8,  4 },
    {  3, 15,  0,  6, 10,  1, 13,  8,  9,  4,  5, 11, 12,  7,  2, 14 }
  },
  {
    {  2, 12,  4,  1,  7, 10, 11,  6,  8,  5,  3, 15, 13,  0, 14,  9 },
    { 14, 11,  2, 12,  4,  7, 13,  1,  5,  0, 15, 10,  3,  9,  8,  6 },
    {  4,  2,  1, 11, 10, 13,  7,  8, 15,  9, 12,  5,  6,  3,  0, 14 },
    { 11,  8, 12,  7,  1, 14,  2, 13,  6, 15,  0,  9, 10,  4,  5,  3 }
  },
  {
    { 12,  1, 10, 15,  9,  2,  6,  8,  0, 13,  3,  4, 14,  7,  5, 11 },
    { 10, 15,  4,  2,  7, 12,  9,  5,  6,  1, 13, 14,  0, 11,  3,  8 },
    {  9, 14, 15,  5,  2,  8, 12,  3,  7,  0,  4, 10,  1, 13, 11,  6 },
    {  4,  3,  2, 12,  9,  5, 15, 10, 11, 14,  1,  7,  6,  0,  8, 13 }
  },
  {
    {  4, 11,  2, 14, 15,  0,  8, 13,  3, 12,  9,  7,  5, 10,  6,  1 },
    { 13,  0, 11,  7,  4,  9,  1, 10, 14,  3,  5, 12,  2, 15,  8,  6 },
    {  1,  4, 11, 13, 12,  3,  7, 14, 10, 15,  6,  8,  0,  5,  9,  2 },
    {  6, 11, 13,  8,  1,  4, 10,  7,  9,  5,  0, 15, 14,  2,  3, 12 }
  },
  {
    { 13,  2,  8,  4,  6, 15, 11,  1, 10,  9,  3, 14,  5,  0, 12,  7 },
    {  1, 15, 13,  8, 10,  3,  7,  4, 12,  5,  6, 11,  0, 14,  9,  2 },
    {  7, 11,  4,  1,  9, 12, 14,  2,  0,  6, 10, 13, 15,  3,  5,  8 },
    {  2,  1, 14,  7,  4, 10,  8, 13, 15, 12,  9,  0,  3,  5,  6, 11 }
  }
};

// --------- HOST -----------
#define P_SIZE 32
static const int hostP[P_SIZE] =
{
  16,  7, 20, 21,
  29, 12, 28, 17,
   1, 15, 23, 26,
   5, 18, 31, 10,
   2,  8, 24, 14,
  32, 27,  3,  9,
  19, 13, 30,  6,
  22, 11,  4, 25
};

// ---------- HOST -----------
#define PC_1_SIZE 56
static const int hostPC_1[PC_1_SIZE] =
{
  57, 49, 41, 33, 25, 17,  9,
   1, 58, 50, 42, 34, 26, 18,
  10,  2, 59, 51, 43, 35, 27,
  19, 11,  3, 60, 52, 44, 36,
  63, 55, 47, 39, 31, 23, 15,
   7, 62, 54, 46, 38, 30, 22,
  14,  6, 61, 53, 45, 37, 29,
  21, 13,  5, 28, 20, 12,  4
};

// ---------- HOST -----------
#define PC_2_SIZE 48
static int hostPC_2[PC_2_SIZE] =
{
  14, 17, 11, 24,  1,  5,
   3, 28, 15,  6, 21, 10,
  23, 19, 12,  4, 26,  8,
  16,  7, 27, 20, 13,  2,
  41, 52, 31, 37, 47, 55,
  30, 40, 51, 45, 33, 48,
  44, 49, 39, 56, 34, 53,
  46, 42, 50, 36, 29, 32
};

// ---------- HOST -----------
static int hostShifts[16] = {
  1, 1, 2, 2, 2, 2, 2, 2,
  1, 2, 2, 2, 2, 2, 2, 1
};

/* --------- DEVICE ----------
   CUDA Constant Memories
   Total: 3008 bytes
   */
__constant__ int devIP       [IP_SIZE];        /*  256 bytes */
__constant__ int devIP_1   [IP_1_SIZE];        /*  256 bytes */
__constant__ int devE         [E_SIZE];        /*  192 bytes */
__constant__ int devS  [S_SIZE][4][16];        /* 2048 bytes */
__constant__ int devP         [P_SIZE];        /*  128 bytes */
__constant__ long long unsigned devKeys[16];   /*  128 bytes */

/* -------------------- DEVICE --------------------------
 * this function conducts DES xor operation */
__device__
void
DES (long long unsigned *MD);

/* -------------------- DEVICE --------------------------
 * the DES function uses this F function during encryption */
__device__
unsigned int
F (unsigned int c, long long unsigned key);

/* -------------------- DEVICE --------------------------
 * the kernel function for DES */
__global__
void
crypt_kernel (int limit, int elements, long long unsigned *data);

/* --------------------- HOST ----------------------------
 * compute key schedule k1 .. k16 */
void
keySchedule (long long unsigned* keys, long long unsigned key);

/* --------------------- HOST ----------------------------
 * this encryption function has the in file as an input,
 * performs encryption with key, and writes to the out file
 * as the output. */
void
encryption (char *in, char *out, char *key, int devThreads);

/* --------------------- HOST ---------------------------
 * this decryption function has the in file as an input,
 * performs decryption with key, and writes to the out file
 * as the output. */
void
decryption (char *in, char *out, char *key, int devThreads);

/* --------------------- HOST ---------------------------
 * helper function for encryption */
void
crypt_des (char *in, char *out, char *key, bool reverse_key, int devBlocks, int devThreads);

#ifdef TEST
static const long long unsigned check_M    = 0x0123456789ABCDEF;
static const long long unsigned check_C    = 0x85E813540F0AB405;
//static const long long unsigned check_key  = 0x133457799BBCDFF1;
static const long long unsigned check_keys[16] =
{
  0x1B02EFFC70720000,
  0x79AED9DBC9E50000,
  0x55FC8A42CF990000,
  0x72ADD6DB351D0000,
  0x7CEC07EB53A80000,
  0x63A53E507B2F0000,
  0xEC84B7F618BC0000,
  0xF78A3AC13BFB0000,
  0xE0DBEBEDE7810000,
  0xB1F347BA464F0000,
  0x215FD3DED3860000,
  0x7571F59467E90000,
  0x97C5D1FABA410000,
  0x5F43B7F2E73A0000,
  0xBF918D3D3F0A0000,
  0xCB3D8B0E17F50000
};

bool
test(void);
#endif // TEST

enum operation {
  UNSPECIFIED = -1,
#ifdef TEST
  TESTING = 0,
#endif // TEST
  ENCRYPT = 1,
  DECRYPT = 2
};

int
main (int argc, char *argv[])
{
  char *inFile = NULL, *outFile = NULL, *keyFile = NULL;
  char opt = -1;
  int index, mode = UNSPECIFIED;
  int threadNum = -1;

  while ((opt = getopt (argc, argv, "i:o:k:t:")) > 0)
  {
    switch (opt)
    {
      case 'i':
        inFile = optarg;
        break;
      case 'o':
        outFile = optarg;
        break;
      case 'k':
        keyFile = optarg;
        break;
      case 't':
        threadNum = atoi(optarg);
        break;
      default:
        break;
    }
  }

  for (index = optind; index < argc; index++)
  {
    if (mode != UNSPECIFIED)
    {
      fprintf(stderr, "des: operation already specified\n");
      return -1;
    }

    if (mode == UNSPECIFIED &&
        strcmp (argv[index], "encrypt") == 0)
    {
      mode = ENCRYPT;
    }
    else if (mode == UNSPECIFIED &&
             strcmp (argv[index], "decrypt") == 0)
    {
      mode = DECRYPT;
    }
#ifdef TEST
    else if (mode == UNSPECIFIED &&
             strcmp (argv[index], "test") == 0)
    {
      mode = TESTING;
    }
#endif // TEST
    else
    {
      fprintf(stderr, "des: unrecognized operation: %s\n", argv[index]);
      return -1;
    }
  }

  if (! (threadNum && !(threadNum & (threadNum - 1))))
  {
    fprintf(stderr, "des: threads must be power of 2\n");
    return -1;
  }

  if (inFile == NULL || outFile == NULL || threadNum < 0 || threadNum > MAX_THREADS )
  {
 #ifdef TEST
    if (mode != TESTING)
    {
      fprintf(stderr, "des: must specify input file and output file\n");
      fprintf(stderr, "Usage: (encrypt | decrypt | test) -i <input_file> -o <output file> -k <key file> -b <number of blocks> -t <number of threads>\n");
      return -1;
    }
#else
    if (threadNum > MAX_THREADS)
    {
      fprintf(stderr, "des: Maximum thread size is 512\n");
    }
    else
    {
      fprintf(stderr, "des: must specify input file and output file\n");
      fprintf(stderr, "Usage: (encrypt | decrypt) -i <input_file> -o <output file> -k <key file> -b <number of blocks> -t <number of threads>\n");
    }

    return -1;
#endif // TEST
  }

  printf("---------------------------------------\n");
  printf("            Device Status\n");
  printf("---------------------------------------\n");

  int i;
  int deviceNum = 0;
  hipGetDeviceCount(&deviceNum);

  struct hipDeviceProp_t prop;

  for (i = 0; i < deviceNum; i++)
  {
    hipGetDeviceProperties(&prop, i);

    printf("Device %d: %s\n", i, prop.name);
    printf("++++++++++++++++++++++++++++++++++++\n");
    printf("totalGlobalMem     : %lu b\n", prop.totalGlobalMem);
    printf("sharedMemPerBlock  : %lu b\n", prop.sharedMemPerBlock);
    printf("regsPerBlock       : %d \n", prop.regsPerBlock);
    printf("maxThreadsPerBlock : %d \n", prop.maxThreadsPerBlock);
    printf("totalConstMem      : %lu b \n", prop.totalConstMem);
    printf("maxGridSize[1:3]   : %d %d %d\n", prop.maxGridSize[0],
                                              prop.maxGridSize[1],
                                              prop.maxGridSize[2]);
    printf("multiProcessorCount: %d\n", prop.multiProcessorCount);
    printf("------------------------------------\n");
  }

  switch (mode)
  {
    case ENCRYPT:
      encryption (inFile, outFile, keyFile, threadNum);
      break;
    case DECRYPT:
      decryption (inFile, outFile, keyFile, threadNum);
      break;
 #ifdef TEST
    case TESTING:
      if (test ())
      {
        printf("OVERALL TEST PASSED\n");
      }
      else
      {
        printf("OVERALL TEST FAILED\n");
      }
      break;
    default:
      fprintf(stderr, "des: must specify operation (decrypt, encrypt, test)\n");
      return -1;
#else
    default:
      fprintf(stderr, "des: must specify operation (decrypt, encrypt)\n");
      return -1;
#endif
 }
  return 0;
}

__device__
void
DES (long long unsigned* MD)
{
  const long long unsigned lMask     = 0x8000000000000000;

  long long unsigned M   = *MD;
  long long unsigned ip  = 0x0;
  long long unsigned iip = 0x0;
  unsigned int  L   = 0x0;
  unsigned int  R   = 0x0;
  unsigned int  t;

  int k;

  for (k = 0; k < IP_SIZE; k++)
  {
     ip |= ((lMask & (M << (devIP[k] - 1))) >> k);
  }

  R = *((unsigned int*)(&ip));
  L = *(((unsigned int*)(&ip)) + 1);

  for (k = 0; k < 16; k++)
  {
    t = L;
    // L(k+1) = R(k)
    L = R;
    // R(k+1) = L(k) ^ f(R(k), K(k+1))
    R = t ^ F(R, devKeys[k]);
  }

  t = L;
  L = R;
  R = t;

  *((unsigned int*)(&ip)) = R;
  *(((unsigned int*)(&ip)) + 1) = L;

  for (k = 0, iip = 0x0; k < IP_1_SIZE; k++)
  {
     iip |= ((lMask & (ip << (devIP_1[k] - 1))) >> k);
  }

  *MD = iip;

  return;
}

__device__
unsigned int
F (unsigned int c, long long unsigned key)
{
  // constants
  //
  // iMask     : mask for MSB of unsigned int
  // lMask     : mask for MSB of long long unsigned
  // firstMask : mask for first bit of first 6 bits
  // lastMask  : mask for last bit  of first 6 bits
  // midMask   : mask for middle 4 bits of first 6 bits
  const unsigned int       iMask     = 0x80000000;
  const long long unsigned lMask     = 0x8000000000000000;
  const long long unsigned firstMask = 0x8000000000000000;
  const long long unsigned lastMask  = 0x0400000000000000;
  const long long unsigned midMask   = 0x7800000000000000;

  // variables
  long long unsigned e   = 0x0; // holds E-expanded key
  long long unsigned c64 = 0x0; // holds c in 64 bit (originally c is 32bit)
  unsigned int       s   = 0x0; // holds S-subtituted key
  unsigned int       p   = 0x0; // holds P-permuted s
  long long unsigned t   = 0x0; // temporary variable
  int i, j, k;                  // temporary variables

  // copy c to c64 (32 bit to 64 bit)
  *((unsigned int*)&c64) = c;
  c64 <<= 32;

  // expand E
  for (k = 0; k < E_SIZE; k++)
  {
    e |= ((lMask & (c64 << (devE[k] - 1))) >> k);
  }

  // XOR with round key
  e = e ^ key;

  // S-Box substitution
  for (k = 0; k < S_SIZE; k++, e <<= 6)
  {
    // get row number
    i = ((firstMask & e) == 0x0) ? 0 : 2;
    i += ((lastMask & e) == 0x0) ? 0 : 1;

    // get column number
    t = (midMask & e) >> 59;
    j = *((int*)&t);

    // merge S
    s |= (devS[k][i][j]) << (4 * (S_SIZE - k - 1));
  }

  // permutation
  for (k = 0; k < P_SIZE; k++)
  {
    p |= ((iMask & (s << (devP[k] - 1))) >> k);
  }

  return p;
}

void
keySchedule (long long unsigned* keys, long long unsigned key)
{
  // constants
  const long long unsigned lMask = 0x8000000000000000;
  const long long unsigned cMask = 0xFFFFFFF000000000;
  const long long unsigned dMask = 0x0000000FFFFFFF00;

  // variables
  long long unsigned C         = 0x0; // holds C0 to C16
  long long unsigned D         = 0x0; // holds D0 to D16
  long long unsigned p         = 0x0; // holds permuted keys
  long long unsigned t         = 0x0; // temporary variable

  // iteration variables
  int k, round;

  // apply key to PC-1
  for (k = 0; k < PC_1_SIZE; k++)
  {
    p |= ((lMask & (key << (hostPC_1[k] - 1))) >> k);
  }

  // compute C0, D0
  C = (p & cMask) | ((p & cMask) >> 28);
  D = (p & dMask) | ((p & dMask) << 28);

  // iterate over round
  for (round = 0; round < 16; round++)
  {
    // shift C(round-1) and D(round-1) to produce C(round), D(round)
    C <<= hostShifts[round];
    D <<= hostShifts[round];

    // merge C and D to apply PC-2
    t = (C & cMask) | ((D & cMask) >> 28);

    // apply PC-2 to produce key(round)
    for (k = 0, p = 0x0; k < PC_2_SIZE; k++)
    {
      p |= ((lMask & (t << (hostPC_2[k] - 1))) >> k);
    }
    keys[round] = p;
  }
}

// IO helpers for encryption/decryption
void
writefile_helper (FILE *fp, long long data[], int num_blocks)
{
  fwrite(data, sizeof(long long), num_blocks, fp);
}

int
readfile_helper (long long unsigned **dst, FILE *fp, unsigned long read_size)
{
  long long unsigned *buffer;
  unsigned long file_read_size;
  unsigned long end, currentPos, delta;
  int blocks;

  currentPos = ftell(fp);
  fseek(fp, 0, SEEK_END);
  end = ftell(fp);
  delta = end - currentPos;
  fseek(fp, -delta, SEEK_CUR);

  if (read_size >= delta)
  {
    // whole file or rest of it can be read in once
    blocks = (int) CEIL(delta, sizeof (long long unsigned));
  }
  else
  {
    // file needs to read multiple times
    blocks = (int) read_size/sizeof(long long unsigned);
    printf("File is split .. reading %d blocks\n", blocks);
  }

  file_read_size = blocks * sizeof(long long unsigned);
  buffer = (long long unsigned*)malloc(file_read_size);

  if (!buffer)
  {
    fprintf(stderr, "dst malloc failed");
    exit(EXIT_FAILURE);
  }

  // Set to null for padding reasons
  if (delta % 8 > 0)
  {
      buffer[blocks-1] = 0;
      if (read_size < delta)
      {
        fprintf(stderr, "this should not happen!");
      }
  }
  fread(buffer, sizeof(long long unsigned), blocks, fp);
  *dst =  buffer;

  return blocks;
}

void
reverse_keys(long long unsigned* keys)
{
    long long keys_tmp[16];
    int i;

    memcpy(keys_tmp, keys, 16 * sizeof(long long));

    for (i = 0; i < 16; i++)
    {
      keys[15-i] = keys_tmp[i];
    }
}

__global__
void
crypt_kernel (int limit, int elements, long long unsigned *data)
{
  int limit2 = (blockIdx.x * blockDim.x + threadIdx.x) * elements + elements;
  for (int index = (blockIdx.x * blockDim.x + threadIdx.x) * elements;
       index < limit2;
       index++)
  {
    if (index < limit)
    {
      DES (&data[index]);
    }
  }
}

void
crypt_des (char *in, char *out, char *key, bool reverse_key, int devThreads)
{
  int NUM_BLOCKS;
  FILE *fp_in, *fp_out, *fp_key;
  long long unsigned *input_data;
  long long unsigned *key_data;
  long long unsigned keys[16];
  long readWriteDataSize;
  struct timeval tstart, tend;
  struct timeval tmemstart, tmemend;
  struct hipDeviceProp_t prop;

  int deviceNum = 0;
  hipGetDeviceCount(&deviceNum);

  // using cuda
  if (deviceNum == 0)
  {
    printf("No CUDA device \n");
    return;
  }
  gettimeofday(&tstart, NULL);

  fp_key = fopen(key, "rb");
  readfile_helper(&key_data, fp_key, 64);
  fclose(fp_key);

  keySchedule(keys, *key_data);
  if (reverse_key)
  {
    reverse_keys(keys);
  }

  hipGetDeviceProperties(&prop, 64);

  size_t totalGlobalMem = prop.totalGlobalMem;
  readWriteDataSize = totalGlobalMem*0.1;
  readWriteDataSize = 1048576000;
  gettimeofday(&tmemstart, NULL);
  // copy constants to device
  hipMemcpyToSymbol (HIP_SYMBOL(devIP), hostIP, sizeof(int)*IP_SIZE);
  hipMemcpyToSymbol (HIP_SYMBOL(devIP_1), hostIP_1, sizeof(int)*IP_1_SIZE);
  hipMemcpyToSymbol (HIP_SYMBOL(devE), hostE, sizeof(int)*E_SIZE);
  hipMemcpyToSymbol (HIP_SYMBOL(devS), hostS, sizeof(int)*S_SIZE*4*16);
  hipMemcpyToSymbol (HIP_SYMBOL(devP), hostP, sizeof(int)*P_SIZE);
  hipMemcpyToSymbol (HIP_SYMBOL(devKeys), keys, sizeof(long long unsigned)*16);

  gettimeofday(&tmemend, NULL);

  printf("Constant mem time: %.5f seconds\n",
       ((double)tmemend.tv_sec + 1.0e-6*tmemend.tv_usec) -
       ((double)tmemstart.tv_sec + 1.0e-6*tmemstart.tv_usec));

  fp_in = fopen(in, "rb");
  fp_out = fopen(out, "wb");
  if (fp_in == NULL || fp_out == NULL)
  {
    fprintf(stderr, "Filepointer failed for %s or %s\n", in, out);
    fclose(fp_in);
    fclose(fp_out);
    exit(EXIT_FAILURE);
  }

  NUM_BLOCKS = readfile_helper(&input_data, fp_in, readWriteDataSize);
  gettimeofday(&tmemstart, NULL);

  while(NUM_BLOCKS > 0)
  {
    printf("NUM_BLOCKS: %d\n",NUM_BLOCKS);
    long long int *output_data = (long long int*)malloc(NUM_BLOCKS*sizeof(long long int));

    // Device data structure
    long long unsigned *devData;       // data array
    // Our problem is 1D
    int devNumBlocks = CEIL(NUM_BLOCKS, devThreads);
    int block_size = 1;


    hipMalloc ((void**)&devData, sizeof(long long int)*NUM_BLOCKS);

    if(CEIL(NUM_BLOCKS, devThreads) > MAX_BLOCKS)
    {
      // We need to increase block size
      printf("MAX BLOCK SIZE REACHED!!!\n");
      block_size = CEIL(CEIL(NUM_BLOCKS, devThreads), MAX_BLOCKS);
      devNumBlocks = CEIL(NUM_BLOCKS, block_size*devThreads);
      printf("Block size: %d devNumBlocks %d total bytes = %d \n",
             block_size, devNumBlocks, block_size*devNumBlocks*devThreads);
    }

    hipMemcpy(devData, input_data, sizeof(long long int)*NUM_BLOCKS, hipMemcpyHostToDevice);

    gettimeofday(&tmemend, NULL);

    printf("Mem time: %.5f seconds\n",
           ((double)tmemend.tv_sec + 1.0e-6*tmemend.tv_usec) -
           ((double)tmemstart.tv_sec + 1.0e-6*tmemstart.tv_usec));

    printf("Blocks %d, Threads per block %d = %d\n",
            devNumBlocks,
            devThreads,
            (devNumBlocks * devThreads));

    crypt_kernel<<<devNumBlocks, devThreads>>>(NUM_BLOCKS, block_size, devData);

    hipMemcpy(output_data, devData, sizeof(long long int)*NUM_BLOCKS, hipMemcpyDeviceToHost);
    hipFree(devData);

    writefile_helper(fp_out, output_data, NUM_BLOCKS);
    gettimeofday(&tend, NULL);
    free(output_data);
    free(input_data);

    NUM_BLOCKS = readfile_helper(&input_data, fp_in, readWriteDataSize);
  }

  fclose(fp_in);
  fclose(fp_out);
  free(key_data);

  printf("Execution time: %.5f seconds\n",
       ((double)tend.tv_sec + 1.0e-6*tend.tv_usec) -
       ((double)tstart.tv_sec + 1.0e-6*tstart.tv_usec));


  return;
}

void
encryption (char *in, char *out, char *key, int devThreads)
{
  printf("des: encryption: in(%s) out(%s), key(%s)\n", in, out, key);
  crypt_des(in, out, key, false, devThreads);

  return;
}

void
decryption (char *in, char *out, char *key, int devThreads)
{
  printf("des: decryption: in(%s) out(%s), key(%s)\n", in, out, key);
  crypt_des(in, out, key, true, devThreads);

  return;
}

#ifdef TEST
bool
test (void)
{
  bool test_flag = true;
  long long unsigned test_keys[16];
  long long unsigned test_M, test_C;

  int i, numBlocks;

  for (i = 0; i < 16; i++)
  {
    test_keys[i] = check_keys[i];
  }

  test_M = check_M;
  numBlocks = 1;

  // copy constants to device
  cudaMemcpyToSymbol (devIP, hostIP, sizeof(int)*IP_SIZE);
  cudaMemcpyToSymbol (devIP_1, hostIP_1, sizeof(int)*IP_1_SIZE);
  cudaMemcpyToSymbol (devE, hostE, sizeof(int)*E_SIZE);
  cudaMemcpyToSymbol (devS, hostS, sizeof(int)*S_SIZE*4*16);
  cudaMemcpyToSymbol (devP, hostP, sizeof(int)*P_SIZE);
  cudaMemcpyToSymbol (devKeys, test_keys, sizeof(long long unsigned)*16);

  // Device data structure
  long long unsigned* devData;
  int *devNumBlocks;

  cudaMalloc ((void**)&devData, sizeof(long long int));
  cudaMalloc ((void**)&devNumBlocks, sizeof(int));

  cudaMemcpy (devData, &test_M, sizeof(long long int), cudaMemcpyHostToDevice);
  cudaMemcpy (devNumBlocks, &numBlocks, sizeof(int), cudaMemcpyHostToDevice);

  printf ("\n");
  printf ("Testing [CUDA DES operation]\n");
  printf ("----------------------------\n");

  crypt_kernel<<<1, 1>>>(devNumBlocks, 1, devData);

  cudaMemcpy (&test_C, devData, sizeof(long long int), cudaMemcpyDeviceToHost);

  if (test_C == check_C)
  {
    printf ("PASSED: DES_encrypt(%llx)=%llx\n", test_M, test_C);
  }
  else
  {
    printf ("FAILED: DES_encrypt(%llx)=%llx, Exp: %llx\n\n", test_M, test_C, check_C);
    test_flag = false;
  }

  reverse_keys(test_keys);
//  memcpy (&test_C, &check_C, 8);

  cudaMemcpyToSymbol (devKeys, test_keys, sizeof(long long unsigned)*16);

  cudaMemcpy (devData, &test_C, sizeof(long long int), cudaMemcpyHostToDevice);

  crypt_kernel<<<1, 1>>>(devNumBlocks, 1, devData);

  cudaMemcpy (&test_M, devData, sizeof(long long int), cudaMemcpyDeviceToHost);

  if (test_M == check_M)
  {
    printf ("PASSED: DES_decrypt(%llx)=%llx\n\n", test_C, test_M);
  }
  else
  {
    printf ("FAILED: DES_decrypt(%llx)=%llx, Exp: %llx\n\n", test_C, test_M, check_M);
    test_flag = false;
  }

  cudaFree (devData);
  cudaFree (devNumBlocks);

  return test_flag;
}
#endif // TEST
